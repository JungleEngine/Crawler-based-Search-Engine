#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <map>
#include <stdio.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

//#define N 5000  // number of nodes
//#define d 0.85  // damping factor used in pageRank algorithm
//#define INITIAL_RANK 1 / double(N)  // initial pageRank value
#define ITERATIONS 10


void logRanks(const float *rank, long nodes)
{
  for(int i = 1; i < nodes; ++i)
  {
    cout << i << " = ";
    cout.precision(5);
    cout << rank[i] << endl;
  }
  cout << endl;
}


__global__ void calculatePageRank(const float *d_rank, float *d_new_rank, const int *d_parents, const int *d_children_count, const int *d_start, const int *d_size, int nodes_num, float d)
{
  // PageRank algorithm formula:
  // PR(A) = (1-d) + d * ( PR(parent_1) / #children(parent_1) + ...) 

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  float new_rank = 0;

  if (i < nodes_num)
  {
    int start_index = d_start[i];
    for (int offset = 0; offset < d_size[i]; ++offset)
    {
      int children_num = d_children_count[d_parents[start_index + offset]];
      if(children_num != 0)
        new_rank += d_rank[d_parents[start_index + offset]] / children_num;
    }

    d_new_rank[i] = (1 - d) + (d * new_rank);
  }
}


int main()
{

  map<string,long> urls;
  map<long,string> reverse_urls;
  map<long,vector<long> > temp_parents;
  map<long,int> temp_children_count;

  string line,value;
  int size;
  long nodes = 1;

  ifstream myfile("web_graph.txt",ios_base::in);

  if (myfile.is_open())
  {
    getline(myfile,line);
    do {
        int i = 2;
        string number = "";
        while(i < line.length() && line[i] != ' ')
        {
            number += line[i++];
        }
        size = stoi(number.c_str());


        getline(myfile,line);
        if(urls.count(line) == 0)
        {
          urls[line] = nodes;
          reverse_urls[nodes++] = line;
        }
        temp_children_count[urls[line]] = size;
        long index = urls[line];


        for(int j = 0 ; j < size ; ++j)
        {
            getline(myfile,value);
            if(urls.count(value) == 0)
            {
              urls[value] = nodes;
              reverse_urls[nodes++] = value;
            }
            temp_parents[urls[value]].push_back(index);
        }

    } while(getline(myfile,line));
      
    myfile.close();
  }

  else cout << "Unable to open file";


    // for (auto i = temp_parents.begin(); i != temp_parents.end(); ++i)
    // {
    //     cout << i->first << " : " << temp_children_count[i->first] << endl;
    //     for(int j = 0 ; j < i->second.size(); ++j)
    //         cout << (i->second)[j] << " ";
    //     cout << endl;
    // }



  int **parents = (int **)malloc(nodes * sizeof(int *));
  int *children_count = (int *)malloc(nodes * sizeof(int));


  long long edges = 0;
  for(int i = 1; i < nodes; ++i)
  {
    if(temp_parents.count(i) != 0)
    {
      parents[i] = (int *)malloc(temp_parents[i].size() * sizeof(int));

      for (int j = 0 ; j < temp_parents[i].size() ; j++)
      {
          parents[i][j] = temp_parents[i][j];
      }
    
      edges = edges + temp_parents[i].size();
    }
    if(temp_children_count.count(i) == 0)
      children_count[i] = 0;
    else children_count[i] = temp_children_count[i];
  }



  float *h_rank = (float *)malloc(sizeof(float) * nodes);
  int *h_parents = (int *)malloc(sizeof(int) * edges);
  int *h_children_count = (int *)malloc(sizeof(int) * nodes);
  int *h_start = (int *)malloc(sizeof(int) * nodes);
  int *h_size = (int *)malloc(sizeof(int) * nodes);

  // Verify that allocations succeeded
    if (h_rank == NULL || h_parents == NULL || h_children_count == NULL || h_start == NULL || h_size == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }


    // Initialize arrays

  for(int i = 1; i < nodes; ++i)
    h_rank[i] = 1 / double(nodes - 1);

  for(int i = 1; i < nodes; ++i)
    h_children_count[i] = children_count[i];

  long long x = 0;
  for(int i = 1 ; i < nodes ;++i)
  {
    h_start[i] = x;
    int j = 0;
    while(j < temp_parents[i].size())
    {
      h_parents[x++] = parents[i][j++];
    }
    h_size[i] = x - h_start[i];
  }



  float *d_rank = NULL;
  float *d_new_rank = NULL;
  int *d_parents = NULL;
  int *d_children_count = NULL;
  int *d_start = NULL;
  int *d_size = NULL;


  hipError_t err = hipSuccess;

  err = hipMalloc((void **)&d_rank, sizeof(float) * nodes);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_rank (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  err = hipMalloc((void **)&d_new_rank, sizeof(float) * nodes);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_new_rank (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  err = hipMalloc((void **)&d_parents, sizeof(int) * edges);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_parents (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  err = hipMalloc((void **)&d_children_count, sizeof(int) * nodes);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_children_count (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  err = hipMalloc((void **)&d_start, sizeof(int) * nodes);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_start (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  err = hipMalloc((void **)&d_size, sizeof(int) * nodes);
  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_size (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }






    printf("Copy input data from the host memory to the CUDA device\n");

    err = hipMemcpy(d_parents, h_parents, sizeof(int) * edges, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector parents from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_children_count, h_children_count, sizeof(int) * nodes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector children_count from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_start, h_start, sizeof(int) * nodes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector start from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_size, h_size, sizeof(int) * nodes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector size from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


  logRanks(h_rank,nodes);


  float d = 0.85;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);



  for (int i = 0 ; i < ITERATIONS ; ++i)
  {

    err = hipMemcpy(d_rank, h_rank, sizeof(float) * nodes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rank from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = ((nodes - 1) / threadsPerBlock) + 1;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);


    // Launch kernel on GPU
    hipEventRecord(start);
    calculatePageRank<<<blocksPerGrid, threadsPerBlock>>>
      (d_rank, d_new_rank, d_parents, d_children_count, d_start, d_size, nodes, d);
    hipEventRecord(stop);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch PageRank kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_rank, d_new_rank, sizeof(float) * nodes, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rank from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    logRanks(h_rank,nodes);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time elapsed in milliseconds: " << milliseconds << endl;
  }



  sort(h_rank + 1, h_rank + nodes, greater<float>());

  ofstream file("ranks.txt");
  if(file.is_open())
  {
    for(int i = 1; i < nodes; ++i)
    {
      file << reverse_urls[i] << endl;
      file.precision(5);
      file << h_rank[i] << endl;
    }
    file.close();
  }
  else cout << "Failed to open file to write!";

  // Free device global memory
  err = hipFree(d_rank);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_new_rank);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_parents);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_children_count);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_start);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_size);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_rank);
  free(h_parents);
  free(h_children_count);
  free(h_start);
  free(h_size);

  printf("Done\n");

  return 0;
}
 